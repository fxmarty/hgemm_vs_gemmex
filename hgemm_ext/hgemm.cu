#include "hip/hip_runtime.h"
#include "hgemm.cuh"

static const char *_cudaGetErrorEnum(hipblasStatus_t error)
{
    switch (error)
    {
        case HIPBLAS_STATUS_SUCCESS:
            return "HIPBLAS_STATUS_SUCCESS";

        case HIPBLAS_STATUS_NOT_INITIALIZED:
            return "HIPBLAS_STATUS_NOT_INITIALIZED";

        case HIPBLAS_STATUS_ALLOC_FAILED:
            return "HIPBLAS_STATUS_ALLOC_FAILED";

        case HIPBLAS_STATUS_INVALID_VALUE:
            return "HIPBLAS_STATUS_INVALID_VALUE";

        case HIPBLAS_STATUS_ARCH_MISMATCH:
            return "HIPBLAS_STATUS_ARCH_MISMATCH";

        case HIPBLAS_STATUS_MAPPING_ERROR:
            return "HIPBLAS_STATUS_MAPPING_ERROR";

        case HIPBLAS_STATUS_EXECUTION_FAILED:
            return "HIPBLAS_STATUS_EXECUTION_FAILED";

        case HIPBLAS_STATUS_INTERNAL_ERROR:
            return "HIPBLAS_STATUS_INTERNAL_ERROR";
    }

    return "<unknown>";
}

void cublas_hgemm
(
    hipblasHandle_t cublas_handle,
    const half* inp,
    const half* w,
    half* out,
    int size_m,
    int size_n,
    int size_k
) {
    const half alpha = __float2half(1.0f);
    const half beta = __float2half(0.0f);

    hipblasStatus_t return_status;

    // PyTorch is row-major while cublas works with column major.
    // Hence Y = A * W in row-major becomes Y^T = W^T * A^T in column major.
    // Hence, m=size_n, n=size_m, k=size_k, and the ordering of `w` and `inp`.
    return_status = hipblasHgemm(cublas_handle,
        HIPBLAS_OP_N,
        HIPBLAS_OP_N,
        size_n, size_m, size_k,
        &alpha,
        w, size_n,
        inp, size_k,
        &beta,
        out, size_n
    );

    if (return_status != HIPBLAS_STATUS_SUCCESS) {
        throw std::runtime_error(_cudaGetErrorEnum(return_status));
    }
}
 
void cublas_gemmex
(
    hipblasHandle_t cublas_handle,
    const half* inp,
    const half* w,
    half* out,
    int size_m,
    int size_n,
    int size_k
) {
    const float alpha = 1.0f;
    const float beta = 0.0f;

    hipblasStatus_t return_status;

    // PyTorch is row-major while cublas works with column major.
    // Hence Y = A * W in row-major becomes Y^T = W^T * A^T in column major.
    // Hence, m=size_n, n=size_m, k=size_k, and the ordering of `w` and `inp`.
    return_status = hipblasGemmEx(cublas_handle,
        HIPBLAS_OP_N,
        HIPBLAS_OP_N,
        size_n, size_m, size_k,
        &alpha,
        w, HIP_R_16F, size_n,
        inp, HIP_R_16F, size_k,
        &beta,
        out, HIP_R_16F, size_n,
        HIPBLAS_COMPUTE_32F, HIPBLAS_GEMM_DEFAULT
    );

    if (return_status != HIPBLAS_STATUS_SUCCESS) {
        throw std::runtime_error(_cudaGetErrorEnum(return_status));
    }
}